#include "hip/hip_runtime.h"
#include "mat_access_pixel.h"

//---------------------CUDA头文件----------------
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <cuda_device_runtime_api.h>
//---------------------CUDA头文件----------------

__global__ void VecAdd(float* A)
{
    int i = threadIdx.x;
    A[i] = A[i] /16;
}
void add(cv::cuda::GpuMat gpuimg)
{
    VecAdd<<<1, 30>>>(gpuimg.data);
}
